#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

__global__ void mykernel(void) {}

int main(void) {
    
    mykernel<<<1, 1>>>();
    printf("Hello World!\n");
    return 0;
}